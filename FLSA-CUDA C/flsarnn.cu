#include <iostream>
#include <cmath>
#include <utility>
#include <flsarnn.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>

#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_real.hpp>
#include <boost/random/variate_generator.hpp>

typedef double value_type;


using namespace std;
using namespace boost::numeric::odeint;

//change this to float if your device does not support double computation


//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
// typedef thrust::host_vector< value_type > state_type;
// typedef thrust::host_vector< size_t > index_vector_type;

void FLSARNN2(float *out, float *y, float *z, float lambda1, float lambda2, int size) {
	float temp;
	float *t = new float[size];

	//Compute t
	for (int i = 0; i < size; i++) {
		if (i == 0)
			temp = y[i] + z[i];
		else if (i == size - 1)
			temp = y[i] - z[i];
		else
			temp = y[i] - z[i - 1] + z[i];

		if (temp > lambda1)
			t[i] = lambda1 - temp;
		else if (temp < -lambda1)
			t[i] = -lambda1 - temp;
		else
			t[i] = 0;
	}

	// Compute z_next
	for (int i = 0; i < size - 1; ++i) {
		temp = z[i] + t[i] - t[i + 1];

		if (temp > lambda2)
			out[i] = lambda2 - z[i];
		else if (temp < -lambda2)
			out[i] = -lambda2 - z[i];
		else
			out[i] = temp - z[i];
	}
}

struct FLSARNN
{
	int N;
	value_type lam1, lam2;
	thrust::device_vector< int > idx;
	thrust::device_vector< value_type > y;

	FLSARNN(value_type lambda1, value_type lambda2, int size, state_type yy)
		: N(size), lam1(lambda1), lam2(lambda2)
	{
		// make new vector of 1, 2, 3 ...
		for (int i = 0; i < size; i++)
		{
			idx.push_back(i);
		}
		y = yy;
	}

	struct FLSARNN_functor1
	{
		value_type lam;
		int N;
		FLSARNN_functor1(value_type lambda, int size)
			: N(size), lam(lambda) {}
		template< class T >
		__host__ __device__
			void operator()(T t) const
		{

			value_type z = thrust::get< 0 >(t);
			value_type zminus1 = thrust::get< 1 >(t);
			value_type y = thrust::get< 2 >(t);
			int indx = thrust::get< 3 >(t);

			value_type temp;
			if (indx == 0)
				temp = y + z;
			else if (indx == N - 1)
				temp = y - z;
			else
				temp = y - zminus1 + z;

			if (temp > lam)
				thrust::get< 4 >(t) = lam - temp;
			else if (temp < -lam)
				thrust::get< 4 >(t) = -lam - temp;
			else
				thrust::get< 4 >(t) = 0.0;
		}
	};

	struct FLSARNN_functor2
	{
		value_type lam;
		FLSARNN_functor2(value_type lambda)
			: lam(lambda) {}
		template< class T >
		__host__ __device__
			void operator()(T t) const
		{
			// unpack the parameter we want to vary and the Lorenz variables
			value_type z = thrust::get< 0 >(t);
			value_type tt = thrust::get< 1 >(t);
			value_type tplus1 = thrust::get< 2 >(t);
			value_type temp = z + tt - tplus1;

			if (temp > lam)
				thrust::get< 3 >(t) = lam - z;
			else if (temp < -lam)
				thrust::get< 3 >(t) = -lam - z;
			else
				thrust::get< 3 >(t) = temp - z;
		}
	};

	template< class State, class Deriv >
	void operator()(const State &x, Deriv &dxdt, value_type t) const
	{
		thrust::device_vector< int > xminus1(N);
		thrust::device_vector< value_type > h(N);
		xminus1.insert(xminus1.begin() + 1, x.begin(), x.end() - 1);

		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					x.begin(),
					xminus1.begin(),
					y.begin(),
					idx.begin(),
					h.begin()
				)),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					x.end(),
					xminus1.end(),
					y.end(),
					idx.end(),
					h.end()
				)),
			FLSARNN_functor1(lam1, N)
		);

		thrust::device_vector< value_type > hplus1(N);
		hplus1.insert(hplus1.begin(), h.begin() + 1, h.end());

		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					x.begin(),
					h.begin(),
					hplus1.begin(),
					dxdt.begin()
				)),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					x.end(),
					h.end(),
					hplus1.end(),
					dxdt.end()
				)),
			FLSARNN_functor2(lam2)
		);
	}
};

int main(int arc, char* argv[])
{
	int driver_version, runtime_version;
	hipDriverGetVersion(&driver_version);
	hipRuntimeGetVersion(&runtime_version);
	cout << driver_version << "\t" << runtime_version << endl;


	state_type x(87);
	vector<value_type> vect{ -0.067885,-0.271394,-0.230591,0.000000,-0.237300,-0.242860,-0.158909,-0.151813,-0.155762,-0.218180,-0.070766,-0.217085,-0.221028,-0.142679,-0.115273,-0.235195,-0.172367,-0.229766,-0.268235,0.000000,-0.103669,-0.063601,-0.143264,-0.175859,-0.210658,-0.114841,-0.198376,-0.176402,-0.132816,-0.173373,0.047142,0.062702,0.145291,-0.051233,-0.032809,0.229869,-0.184480,2.257590,2.387689,0.756727,0.311959,0.131189,0.557582,-0.235412,0.099662,-0.146354,-0.270817,-0.219345,0.004307,0.001176,-0.128079,-0.135935,0.144436,-0.238093,0.517836,0.430683,0.585686,0.693343,0.690585,0.465043,0.237685,0.478356,0.422885,-0.265142,0.079919,0.155776,0.915878,0.085746,-0.050649,0.062327,0.234434,0.084003,-0.054938,0.177567,0.220642,0.130591,0.124412,0.005821,0.066331,0.231169,0.116183,0.236077,0.074800,0.137732,-0.005529,-0.008017,0.217077 };
	//value_type* yy = &vect[0];
	thrust::device_vector< value_type > y(87);// { -0.067885, -0.271394, -0.230591, 0.000000, -0.237300, -0.242860, -0.158909, -0.151813, -0.155762, -0.218180, -0.070766, -0.217085, -0.221028, -0.142679, -0.115273, -0.235195, -0.172367, -0.229766, -0.268235, 0.000000, -0.103669, -0.063601, -0.143264, -0.175859, -0.210658, -0.114841, -0.198376, -0.176402, -0.132816, -0.173373, 0.047142, 0.062702, 0.145291, -0.051233, -0.032809, 0.229869, -0.184480, 2.257590, 2.387689, 0.756727, 0.311959, 0.131189, 0.557582, -0.235412, 0.099662, -0.146354, -0.270817, -0.219345, 0.004307, 0.001176, -0.128079, -0.135935, 0.144436, -0.238093, 0.517836, 0.430683, 0.585686, 0.693343, 0.690585, 0.465043, 0.237685, 0.478356, 0.422885, -0.265142, 0.079919, 0.155776, 0.915878, 0.085746, -0.050649, 0.062327, 0.234434, 0.084003, -0.054938, 0.177567, 0.220642, 0.130591, 0.124412, 0.005821, 0.066331, 0.231169, 0.116183, 0.236077, 0.074800, 0.137732, -0.005529, -0.008017, 0.217077 };

	// initialize x
	thrust::fill(x.begin(), x.begin(), 0);
	y.insert(y.begin(), vect.begin(), vect.end());
	//y.insert(y.begin(), *yy, *(yy + 87));

	FLSARNN flsa(0.5, 0.2, 87, y);

	// create stepper
	runge_kutta4< state_type, value_type, state_type, value_type > stepper;

	int step = integrate(flsa, x, 0.0, 10.0, 0.1);
	std::cout << step << std::endl;
	thrust::copy(x.begin(), x.end(), std::ostream_iterator< value_type >(std::cout, "\n"));
	std::cout << std::endl;

	return 0;
}
void CopyData(state_type des, float *src, int N)
{
	for (int i = 0; i < N; i++)
	{
		des[i] = src[i];
	}
}
void flsarnn(float *out, float *y, float *z, float* lambda, int size)
{
	value_type lambda1 = lambda[0], lambda2 = lambda[1];
	state_type x(size), yy(size);
	CopyData(x, z, size);
	CopyData(yy, y, size);
	FLSARNN flsa(lambda1, lambda2, size, yy);
	integrate(flsa, x, 0.0, 10.0, 0.1);
	for (int i = 0; i < size; i++)
	{
		out[i] = x[i];
	}
}
